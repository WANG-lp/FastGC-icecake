#include "hip/hip_runtime.h"
#include "cuda.cuh"
#include <hip/hip_runtime.h>

__global__ void _cusub(int *a, int *b, int *c) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *c = *a - *b;
    }
}

int sub_cuda(int a, int b) {
    int *d_a, *d_b, *d_c;
    int out = 0;

    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));
    hipMalloc(&d_c, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    _cusub<<<1, 1>>>(d_a, d_b, d_c);

    hipMemcpy(&out, d_c, sizeof(int), hipMemcpyDeviceToHost);
    return out;
}